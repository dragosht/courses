#include "hip/hip_runtime.h"
// MP 5 Scan
// Given a list (lst) of length n
// Output its prefix sum = {lst[0], lst[0] + lst[1], lst[0] + lst[1] + ... + lst[n-1]}

#include    <wb.h>

#define BLOCK_SIZE 512 //@@ You can change this
#define SECTION_SIZE BLOCK_SIZE

#define wbCheck(stmt) do {                                 \
        hipError_t err = stmt;                            \
        if (err != hipSuccess) {                          \
            wbLog(ERROR, "Failed to run stmt ", #stmt);    \
            return -1;                                     \
        }                                                  \
    } while(0)


__global__ void adjust(float* output, float* sums, int len) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < len && blockIdx.x > 0) {
	output[i] += sums[blockIdx.x - 1];
    }
}


__global__ void scan(float * input, float * output, float* sums, int len) {
    //@@ Modify the body of this function to complete the functionality of
    //@@ the scan on the device
    //@@ You may need multiple kernel calls; write your kernels before this
    //@@ function and call them from here

    __shared__ float xy[SECTION_SIZE];
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < len) {
	xy[threadIdx.x] = input[i];
    }

    for (unsigned int stride = 1; stride <= threadIdx.x; stride *= 2) {
	__syncthreads();
	float in1 = 0.0;
	if (threadIdx.x >= stride) {
	    in1 = xy[threadIdx.x - stride];
	}
	__syncthreads();
	xy[threadIdx.x] += in1;
    }

    __syncthreads();

    if (i < len) {
	output[i] = xy[threadIdx.x];
    }

    if (sums != NULL) {
	if (threadIdx.x == blockDim.x - 1) {
	    sums[blockIdx.x] = output[i];
	}
    }
}

int main(int argc, char ** argv) {
    wbArg_t args;
    float * hostInput; // The input 1D list
    float * hostOutput; // The output list
    float * deviceInput;
    float * deviceOutput;
    float * deviceSums;
    float * deviceSums2;
    int numElements; // number of elements in the list

    args = wbArg_read(argc, argv);

    wbTime_start(Generic, "Importing data and creating memory on host");
    hostInput = (float *) wbImport(wbArg_getInputFile(args, 0), &numElements);
    hostOutput = (float*) malloc(numElements * sizeof(float));
    wbTime_stop(Generic, "Importing data and creating memory on host");

    wbLog(TRACE, "The number of input elements in the input is ", numElements);

    wbTime_start(GPU, "Allocating GPU memory.");
    wbCheck(hipMalloc((void**)&deviceInput, numElements*sizeof(float)));
    wbCheck(hipMalloc((void**)&deviceOutput, numElements*sizeof(float)));
    wbCheck(hipMalloc((void**)&deviceSums, (numElements/BLOCK_SIZE + 1) * sizeof(float)));
    wbCheck(hipMalloc((void**)&deviceSums2, (numElements/BLOCK_SIZE + 1) * sizeof(float)));
    wbTime_stop(GPU, "Allocating GPU memory.");

    wbTime_start(GPU, "Clearing output memory.");
    wbCheck(hipMemset(deviceOutput, 0, numElements*sizeof(float)));
    wbTime_stop(GPU, "Clearing output memory.");

    wbTime_start(GPU, "Copying input memory to the GPU.");
    wbCheck(hipMemcpy(deviceInput, hostInput, numElements*sizeof(float), hipMemcpyHostToDevice));
    wbTime_stop(GPU, "Copying input memory to the GPU.");

    //@@ Initialize the grid and block dimensions here
    int numBlocks = (numElements - 1) / BLOCK_SIZE + 1;

    wbTime_start(Compute, "Performing CUDA computation");
    //@@ Modify this to complete the functionality of the scan
    //@@ on the deivce
    dim3 dimGrid1(numBlocks, 1, 1);
    dim3 dimBlock1(BLOCK_SIZE, 1, 1);
    scan<<<dimGrid1, dimBlock1>>>(deviceInput, deviceOutput, deviceSums, numElements);
    wbCheck(hipDeviceSynchronize());

    dim3 dimGrid2(1, 1, 1);
    dim3 dimBlock2(numBlocks, 1, 1);
    scan<<<dimGrid2, dimBlock2>>>(deviceSums, deviceSums2, NULL, numBlocks);
    wbCheck(hipDeviceSynchronize());


    dim3 dimGrid3(numBlocks, 1, 1);
    dim3 dimBlock3(BLOCK_SIZE, 1, 1);
    adjust<<<dimGrid3, dimBlock3>>>(deviceOutput, deviceSums2, numElements);
    wbCheck(hipDeviceSynchronize());

    wbTime_stop(Compute, "Performing CUDA computation");

    wbTime_start(Copy, "Copying output memory to the CPU");
    wbCheck(hipMemcpy(hostOutput, deviceOutput, numElements*sizeof(float), hipMemcpyDeviceToHost));
    wbTime_stop(Copy, "Copying output memory to the CPU");

    wbTime_start(GPU, "Freeing GPU Memory");
    wbCheck(hipFree(deviceInput));
    wbCheck(hipFree(deviceOutput));
    wbCheck(hipFree(deviceSums));
    wbCheck(hipFree(deviceSums2));
    wbTime_stop(GPU, "Freeing GPU Memory");

    wbSolution(args, hostOutput, numElements);

    free(hostInput);
    free(hostOutput);

    return 0;
}

