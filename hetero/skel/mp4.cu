#include "hip/hip_runtime.h"
// MP 4 Reduction
// Given a list (lst) of length n
// Output its sum = lst[0] + lst[1] + ... + lst[n-1];

#include    <wb.h>

#define BLOCK_SIZE 512 //@@ You can change this

#ifndef wbCheck
#define wbCheck(stmt) do {                                 \
        hipError_t err = stmt;                            \
        if (err != hipSuccess) {                          \
            wbLog(ERROR, "Failed to run stmt ", #stmt);    \
            return -1;                                     \
        }                                                  \
} while(0)
#endif //wbCheck


__global__ void total(float * input, float * output, int len) {
    //@@ Load a segment of the input vector into shared memory
    //@@ Traverse the reduction tree
    //@@ Write the computed sum of the block to the output vector at the
    //@@ correct index
    __shared__ float partSum[2 * BLOCK_SIZE];
    unsigned int b = blockIdx.x;
    unsigned int t = threadIdx.x;
    unsigned int start = 2 * blockIdx.x * blockDim.x;

    if (start + t < len) {
        partSum[t] = input[start + t];
    } else {
	partSum[t] = 0.0;
    }

    if (start + blockDim.x + t < len) {
        partSum[blockDim.x + t] = input[start + blockDim.x + t];
    } else {
	partSum[blockDim.x + t] = 0.0;
    }

    //The inefficient version
    /*
    for (unsigned int stride = 1; stride <= blockDim.x; stride *= 2) {
	__syncthreads();
	if (t % stride == 0) {
	    partSum[2 * t] += partSum[2 * t + stride];
	}
    }
    */

    //The better version
    for (unsigned int stride = blockDim.x; stride > 0; stride /= 2) {
	__syncthreads();
	if (t < stride) {
	    partSum[t] += partSum[t + stride];
	}
    }

    if (t == 0) {
	output[b] = partSum[0];
    }
}

int main(int argc, char ** argv) {
    int ii;
    wbArg_t args;
    float * hostInput; // The input 1D list
    float * hostOutput; // The output list
    float * deviceInput;
    float * deviceOutput;
    int numInputElements; // number of elements in the input list
    int numOutputElements; // number of elements in the output list

    args = wbArg_read(argc, argv);

    wbTime_start(Generic, "Importing data and creating memory on host");
    hostInput = (float *) wbImport(wbArg_getInputFile(args, 0), &numInputElements);

    numOutputElements = numInputElements / (BLOCK_SIZE << 1);
    if (numInputElements % (BLOCK_SIZE<<1)) {
        numOutputElements++;
    }
    hostOutput = (float*) malloc(numOutputElements * sizeof(float));

    wbTime_stop(Generic, "Importing data and creating memory on host");

    wbLog(TRACE, "The number of input elements in the input is ", numInputElements);
    wbLog(TRACE, "The number of output elements in the input is ", numOutputElements);

    wbTime_start(GPU, "Allocating GPU memory.");
    //@@ Allocate GPU memory here
    wbCheck(hipMalloc((void**) &deviceInput, numInputElements * sizeof(float)));
    wbCheck(hipMalloc((void**) &deviceOutput, numOutputElements * sizeof(float)));

    wbTime_stop(GPU, "Allocating GPU memory.");

    wbTime_start(GPU, "Copying input memory to the GPU.");
    //@@ Copy memory to the GPU here
    wbCheck(hipMemcpy(deviceInput, hostInput, numInputElements * sizeof(float), hipMemcpyHostToDevice));

    wbTime_stop(GPU, "Copying input memory to the GPU.");
    //@@ Initialize the grid and block dimensions here

    wbTime_start(Compute, "Performing CUDA computation");
    //@@ Launch the GPU Kernel here
    dim3 dimGrid((numInputElements - 1) / BLOCK_SIZE + 1, 1, 1);
    dim3 dimBlock(BLOCK_SIZE, 1, 1);

    for (ii = 0; ii < numOutputElements; ii++) {
	total<<<dimGrid, dimBlock>>>(deviceInput, deviceOutput, numInputElements);
    }

    wbCheck(hipDeviceSynchronize());
    wbTime_stop(Compute, "Performing CUDA computation");

    wbTime_start(Copy, "Copying output memory to the CPU");
    //@@ Copy the GPU memory back to the CPU here
    wbCheck(hipMemcpy(hostOutput, deviceOutput, numOutputElements * sizeof(float), hipMemcpyDeviceToHost));

    wbTime_stop(Copy, "Copying output memory to the CPU");

    /********************************************************************
     * Reduce output vector on the host
     * NOTE: One could also perform the reduction of the output vector
     * recursively and support any size input. For simplicity, we do not
     * require that for this lab.
     ********************************************************************/
    for (ii = 1; ii < numOutputElements; ii++) {
        hostOutput[0] += hostOutput[ii];
    }

    wbTime_start(GPU, "Freeing GPU Memory");
    //@@ Free the GPU memory here
    wbCheck(hipFree(deviceInput));
    wbCheck(hipFree(deviceOutput));

    wbTime_stop(GPU, "Freeing GPU Memory");

    wbSolution(args, hostOutput, 1);

    free(hostInput);
    free(hostOutput);

    return 0;
}

