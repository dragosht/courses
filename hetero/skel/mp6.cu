#include "hip/hip_runtime.h"
#include    <wb.h>

#define wbCheck(stmt) do {                                 \
        hipError_t err = stmt;                            \
        if (err != hipSuccess) {                          \
            wbLog(ERROR, "Failed to run stmt ", #stmt);    \
            return -1;                                     \
        }                                                  \
    } while(0)


#define MASK_WIDTH  5
#define MASK_RADIUS 2

//@@ INSERT CODE HERE
#define TILE_WIDTH 16
#define SHARED_SIZE TILE_WIDTH + 2 * MASK_RADIUS

__device__ float clamp(float x, float start, float end)
{
    return (min(max(x, start), end));
}


__global__ void convolute(float* in, float* out, int height, int width,
			  int channels, const float *M)
{
    __shared__ float Ns[SHARED_SIZE][SHARED_SIZE];

    const int corners[] = {-MASK_RADIUS, MASK_RADIUS};
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int row = by * blockDim.y + ty;
    int col = bx * blockDim.x + tx;

    if (row >= height || col >= width) {
	return;
    }

    for (int k = 0; k < channels; k++) {
	    for (int i = 0; i < 2; i++) {
	    for (int j = 0; j < 2; j++) {
		int x = corners[i];
		int y = corners[j];
		if (row + y < 0 || row + y >= height ||
		    col + x < 0 || col + x >= width) {
		    Ns[ty + y + MASK_RADIUS][tx + x + MASK_RADIUS] = 0.;
		} else {
		    Ns[ty + y + MASK_RADIUS][tx + x + MASK_RADIUS] =
			in[((row + y)* width + (col + x)) * channels + k];
		}
	    }
	}

	__syncthreads();
	float accum = 0.0;
	for (int y = -MASK_RADIUS; y <= MASK_RADIUS; y++) {
	    for (int x = -MASK_RADIUS; x <= MASK_RADIUS; x++) {
		int yOffset = row + y;
		int xOffset = col + x;

		if (xOffset >= 0 && xOffset < width &&
		    yOffset >= 0 && yOffset < height) {
		    float value = Ns[ty + y + MASK_RADIUS][tx + x + MASK_RADIUS];
		    //float value = in[((row + y)* width + (col + x)) * channels + k];
		    float mask = M[(y + MASK_RADIUS) * MASK_WIDTH + x + MASK_RADIUS];
		    accum += value * mask;
		}

	    }
	}

	out[(row * width + col) * channels + k] = clamp(accum, 0., 1.);
	__syncthreads();
    }
}


int main(int argc, char* argv[]) {
    wbArg_t arg;
    int maskRows;
    int maskColumns;
    int imageChannels;
    int imageWidth;
    int imageHeight;
    char * inputImageFile;
    char * inputMaskFile;
    wbImage_t inputImage;
    wbImage_t outputImage;
    float * hostInputImageData;
    float * hostOutputImageData;
    float * hostMaskData;
    float * deviceInputImageData;
    float * deviceOutputImageData;
    float * deviceMaskData;

    arg = wbArg_read(argc, argv); /* parse the input arguments */

    inputImageFile = wbArg_getInputFile(arg, 0);
    inputMaskFile = wbArg_getInputFile(arg, 1);

    inputImage = wbImport(inputImageFile);
    hostMaskData = (float *) wbImport(inputMaskFile, &maskRows, &maskColumns);

    assert(maskRows == 5); /* mask height is fixed to 5 in this mp */
    assert(maskColumns == 5); /* mask width is fixed to 5 in this mp */

    imageWidth = wbImage_getWidth(inputImage);
    imageHeight = wbImage_getHeight(inputImage);
    imageChannels = wbImage_getChannels(inputImage);

    outputImage = wbImage_new(imageWidth, imageHeight, imageChannels);

    hostInputImageData = wbImage_getData(inputImage);
    hostOutputImageData = wbImage_getData(outputImage);

    wbTime_start(GPU, "Doing GPU Computation (memory + compute)");

    wbTime_start(GPU, "Doing GPU memory allocation");
    hipMalloc((void **) &deviceInputImageData, imageWidth * imageHeight * imageChannels * sizeof(float));
    hipMalloc((void **) &deviceOutputImageData, imageWidth * imageHeight * imageChannels * sizeof(float));
    hipMalloc((void **) &deviceMaskData, maskRows * maskColumns * sizeof(float));
    wbTime_stop(GPU, "Doing GPU memory allocation");


    wbTime_start(Copy, "Copying data to the GPU");
    hipMemcpy(deviceInputImageData,
               hostInputImageData,
               imageWidth * imageHeight * imageChannels * sizeof(float),
               hipMemcpyHostToDevice);
    hipMemcpy(deviceMaskData,
               hostMaskData,
               maskRows * maskColumns * sizeof(float),
               hipMemcpyHostToDevice);
    wbTime_stop(Copy, "Copying data to the GPU");


    wbTime_start(Compute, "Doing the computation on the GPU");
    //@@ INSERT CODE HERE
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);
    dim3 dimGrid((wbImage_getWidth(inputImage) - 1) / TILE_WIDTH + 1,
		 (wbImage_getHeight(inputImage) - 1) / TILE_WIDTH + 1,
		 1);
    convolute<<<dimGrid, dimBlock>>>(deviceInputImageData, deviceOutputImageData, imageHeight, imageWidth,
		imageChannels, deviceMaskData);

    wbTime_stop(Compute, "Doing the computation on the GPU");


    wbTime_start(Copy, "Copying data from the GPU");
    hipMemcpy(hostOutputImageData,
               deviceOutputImageData,
               imageWidth * imageHeight * imageChannels * sizeof(float),
               hipMemcpyDeviceToHost);
    wbTime_stop(Copy, "Copying data from the GPU");

    wbTime_stop(GPU, "Doing GPU Computation (memory + compute)");

    wbSolution(arg, outputImage);

    hipFree(deviceInputImageData);
    hipFree(deviceOutputImageData);
    hipFree(deviceMaskData);

    free(hostMaskData);
    wbImage_delete(outputImage);
    wbImage_delete(inputImage);

    return 0;
}
